#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>
#include <iostream>

#include "config.h"
//#include "cuPrintf.cuh"
#include "reduction.h"

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

using namespace std;

typedef float Doub;
typedef float Float;
typedef int Int;

__device__ float PI=3.141592653589793f;

__device__ bool _debug0 = false;
bool debug0_dev = false;

// global device variables for x_table kernel
bool gmem_x_table_is_initialised = false;                   // has global dev memory already been allocated
float *d_list_all;
float *d_particles;

// global device variables for accel kernel
bool gmem_accel_is_initialised = false;                     // has global dev memory already been allocated
float *d_accels;

// tables with constant values for accel kernel
__constant__ float x_constants[X_TABLE_ENTRIES];
__constant__ float gamma1_constants[GAMMA1_TABLE_ENTRIES];
__constant__ float gamma2_constants[GAMMA2_TABLE_ENTRIES];
//__device__ float buggyval[50];

// Function declarations
unsigned int nextPow2( unsigned int x );
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads);
__device__ float _ultraspher(float xi, int n, float alpha);
__device__ void _print_cache(float *cache);
__device__ void _print_list_all(float *list_all);
__global__ void print_cuda_array(float *arr, int len, int whichrow);
void x_table_cleanup();
void accel_cleanup();

//***********************************************************
// index mapper functions
//***********************************************************
// this map groups the entries with the same (n,x,l,m) indices
// but different particle (threadidx) indices
__device__ inline int _map_cache(int n, int threadidx, int x, int l, int m)
{
    const int N_lm = TRIANGLE_ENTRIES;
    int idx = n*N_lm + x*(N_lm/2) + (l*(l+1))/2 + m;
    idx *= BLOCK_DIM_X;
    idx += threadidx;
    return idx;
}
// some other possible layout for the cache
__device__ inline int _map_cache2(int n, int threadidxx, int x, int l, int m)
{
    const int N_lm = TRIANGLE_ENTRIES;
    int idx = N_lm*(threadidxx + n*BLOCK_DIM_X);
    idx += x*(N_lm/2) + l*(l+1)/2 + m;
    return idx;
}
// map to list-all
// optimized for coalesced access: entries with the same indices
// (n,x,l,m) come in a row (i.e. one after each other)
__device__ inline int _map_list_all(int grid, int block, int n, int x, int l, int m)
{
    const int N_lm = TRIANGLE_ENTRIES;
    int idx = n*N_lm + x*(N_lm/2) + (l*(l+1))/2 + m;
    idx *= GRID_DIM_X*NUM_GRIDS;
    idx += GRID_DIM_X*grid;
    idx += block;
    return idx;
}
// map to X-table
__device__ int _map_x_table(int n, int x, int l, int m)
{
    const int N_lm = TRIANGLE_ENTRIES;
    int idx = n*N_lm + x*(N_lm/2) + l*(l+1)/2 + m;
    return idx;
}
int map_x_table(int n, int x, int l, int m)
{
    const int N_lm = TRIANGLE_ENTRIES;
    int idx = n*N_lm + x*(N_lm/2) + l*(l+1)/2 + m;
    return idx;
}
// map to gamma1-table
__device__ int _map_gamma1_table(int n, int l)
{
    int idx = n*N_L + l;
    return idx;
}
int map_gamma1_table(int n, int l)
{
    int idx = n*N_L + l;
    return idx;
}

//***********************************************************
// This kernel evaluates the values of the individual terms
// and stores the resulting (reduced) table of each block
// to d_list_all
// Args:
// d_list_all: a 2D-array to store the list of tables
// Rem: do not use for timing
//***********************************************************
__global__ void _compute_all_list(float *particles, int nparticles, float *d_list_all)
{
    const int n = threadIdx.y;								// this threads n-index
                                                            // within the block
    const int block = blockIdx.x;							// this threads block-index
    const int tid = threadIdx.x;
    __shared__ float cache[CACHE_ENTRIES];                  // size = number of threads * triangle size

    int p = block*BLOCK_DIM_X + threadIdx.x;                // particle which is processed by this thread
    int grid = 0;

    // forward declarations
    float fact, fact2, oldfact,pll,pmm, _pmm, pmmp1,omx2;
    float sign;
    float P_l_m;
    float inv_renorm;                                       // inverse renormalization factor in (6.7.8)
    // internal tables
    float cmphi[MAX_L+1];                                   // cos(m*phi_k)
    float smphi[MAX_L+1];                                   // sin(m*phi_k)
    float chi[MAX_L+1];                                     // prefactor of (Hernquist 2.25)
    float pi_4 = 4.0f*PI;
    // registers
    float rk, thk, phik, mk, xi_k, x;
    float C_n_k, phi_nl_t_k, term_0, term_1, new_m;
    float ny_0_0, ny_m_m, ny_l_m;
    float chi_0, chi_l, fac_chi;
    int ll;

    while(p < nparticles)
    {
        // step 1: load the data from global memory,
        // compute from this data the terms to sum up
        // and store the values in the cache

        // load the data from global memory
        rk = particles[N_DOFS*p];
        thk = particles[N_DOFS*p + 1];	// theta_k
        phik = particles[N_DOFS*p + 2];	// phi_k
        mk = particles[N_DOFS*p + 6];		// mass of particle k
        //mk = 1.0;
        xi_k = (rk - 1.0f) / (rk + 1.0f);
        x = cos(thk);
        omx2=(1.0f-x)*(1.0f+x);
        // build the sincos table
        for(int m = 0; m <= MAX_L; m++)
        {
            __sincosf(m*phik, &smphi[m], &cmphi[m]);           // accurate?
            //cmphi[m] = cos(m*phik);
            //smphi[m] = sin(m*phik);
        }
        // build the chi table
        chi_0 = 1.0f/(rk + 1.0f);
        chi_l = chi_0;
        fac_chi = rk*chi_0*chi_0;
        for(int l = 0; l <= MAX_L; l++)
        {
            chi[l] = chi_l;
            chi_l *= fac_chi;
        }

        // init recursions
        sign = 1.0;             // sign in (NR3 6.7.10)
        ny_0_0 = 1.0f;           // ny init (inverse renormalization for
        ny_m_m = ny_0_0;        // legendre polynomials, see NR3)
        _pmm=1.0f;               // legendre init (underscore means no prefactor)
        fact2=1.0f;
        for(int m = 0; m <= MAX_L; m++)
        {
            // compute pmm (with prefactor, see (6.7.10) )
            pmm=sqrt((2.0f*m+1.0f)*_pmm/(pi_4));
            pmm *= sign;
            sign *= -1.0f;
            // compute P_m+1_m
            pmmp1=x*sqrt(2.0f*m+3.0f)*pmm;
            oldfact = sqrt(2.0f*m+3.0f);
            // compute inverse renormalization
            // factor (diagonal entry of lm-triangle)
            ny_l_m = ny_m_m;

            for(int l = m; l <= MAX_L; l++)
            {
                // compute P_l_m
                inv_renorm = (pi_4)/(2.0f*l+1.0f);                          // accurate ?
                inv_renorm *= ny_l_m;
                inv_renorm = sqrt(inv_renorm);
                P_l_m = pmm * inv_renorm;

                // compute other quantities
                C_n_k = _ultraspher(xi_k, n, 2.0f*l + 1.5f);                // accurate ?
                phi_nl_t_k = - chi[l] * C_n_k * sqrt(pi_4);

                // now, compute the final terms and
                // store them in cache
                term_0 = mk * phi_nl_t_k * P_l_m * cmphi[m];
                term_1 = mk * phi_nl_t_k * P_l_m * smphi[m];
                int idx_0 = _map_cache(n, tid, 0, l, m);
                int idx_1 = _map_cache(n, tid, 1, l, m);
                cache[idx_0] = term_0;
                cache[idx_1] = term_1;
                //if(n==1 && l == 0 && m == 0)
                //{
                 //   cuPrintf("GPU (k=%d n=%d x=%d l=%d m=%d): %f\n", p, n, 0, l, m, term_0);
                //}

                // update inverse renormalization factor
                ny_l_m *= ((l+m+1.0f)/(l-m+1.0f) );

                // update legendre polynomial
                ll = l + 2;
                fact=sqrt((4.0f*ll*ll-1.0f)/(ll*ll-m*m));                // accurate?
                pll=(x*pmmp1-pmm/oldfact)*fact;
                oldfact=fact;
                pmm=pmmp1;
                pmmp1=pll;
            }

            // update inverse renormalization factor
            new_m = m+1;
            ny_m_m *= (2*new_m-1)*(2*new_m);
            // update legendre polynomial
            _pmm *= omx2*fact2/(fact2+1.0f);
            fact2 += 2.0f;
        }

        __syncthreads();

        // print the cache
        #ifdef GPU_PRINT
        /*cuPrintfRestrict(0,CUPRINTF_UNRESTRICTED);
        cuPrintf("original cache, block %d: \n", block);
        _print_cache(cache);
        cuPrintfRestrict(CUPRINTF_UNRESTRICTED, CUPRINTF_UNRESTRICTED);*/
        #endif

        // step 2: reduction of current block with non-divergent warps
        // (sequential adressing)
        // and swapped loop order
        for(int x = 0; x < 2; x++)
        {
            for(int l = 0; l <= MAX_L; l++)
            {
                for (int m = 0; m <= MAX_L; m++)
                {
                    //cuPrintf("reducing at block %d: (%d %d %d)\n", blockIdx.x, x, l, m);
                    if(m <= l)
                    {
                        // reduce all the tid's of the current
                        // index n-tuple (n,x,l,m)
                        for(unsigned int s=blockDim.x/2; s > 0; s>>=1)
                        {
                            if(tid < s)
                            {
                                int idx1 = _map_cache(n,tid,x,l,m);
                                int idx2 = _map_cache(n, tid+s, x, l, m);
                                cache[idx1] += cache[idx2];
                            }
                            // __syncthreads();                      // TODO: can probably be omitted, since each
                                                                    // row consists in exactly one warp
                        }
                    }
                }
            }
        }

        __syncthreads();

        // print the cache
        #ifdef GPU_PRINT
        /*cuPrintfRestrict(0,CUPRINTF_UNRESTRICTED);
        cuPrintf("reduced cache, block %d: \n", block);
        _print_cache(cache);
        cuPrintfRestrict(CUPRINTF_UNRESTRICTED, CUPRINTF_UNRESTRICTED);*/
        #endif

        // step 3: write the reduction of the current block to the
        // all list, which resides in global memory
        if(tid == 0)                            // reduced table is in first thread of each block
        {
            for(int x = 0; x < 2; x++)
            {
                for(int l = 0; l <= MAX_L; l++)
                {
                    for (int m = 0; m <= MAX_L; m++)
                    {
                        int list_all_idx = _map_list_all(grid, block, n, x, l, m);
						int cache_idx = _map_cache(n, tid, x, l, m);
                        if(m <= l)
                        {
                            d_list_all[list_all_idx] = cache[cache_idx];
                        }
                    }
                }
            }
        }

        // step 4: move to the next grid
        grid = grid + 1;
        p += GRID_DIM_X*BLOCK_DIM_X;

        __syncthreads();
    } // while (p)
}

//***********************************************************
// This kernel computes a list of the accelerations acting
// on each particles
// to d_list_all
// Args:
// particles, nparticles: the particle data
// x_table: table containing the x1- and x2-table from
//          the host code
// gamma_table: the gamma table
// Result:
// compute the accelerations and store them in accels
//***********************************************************
__global__ void _accel(float *particles, int nparticles, float *accels)
{
    const int block = blockIdx.x;							// this threads block-index
    const int tid = threadIdx.x;
    int p = block*ACCEL_BLOCK_DIM_X + threadIdx.x;                // particle which is processed by this thread
    int grid = 0;

    while(p < nparticles)
    {
        // step 1: load the data from global memory,
        // and then compute and sum up the terms

        // load the data from global memory
        float r = particles[p*N_DOFS];
        float th = particles[p*N_DOFS + 1];
        float phi = particles[p*N_DOFS + 2];
        float x = cos(th);
        float a_x, a_y, a_z;
        int maxl = MAX_L;
        int maxn = MAX_N;
        float a_r = 0, a_th = 0, a_phi = 0;

        // forward declarations for recurrence relation of legendre polynomials
        float fact, fact2, oldfact,pll,pmm, _pmm, pmmp1,omx2;
        float P_l_m, P_lm1_m, P_m_m;
        float DP_m_m, DP_l_m;
        float inv_renorm;          // inverse renormalization factor in (6.7.8)
        float sign;
        float new_m;
        float ny_0_0, ny_m_m, ny_l_m;
        float theta_m_m, theta_l_m; // recursion for (l-m)!/(l+m)!
        theta_m_m = 1.0f;
        int ll;
        omx2=(1.0f-x)*(1.0f+x);
        float pi_4 = 4.0f*PI;
        // inverse renormalization for legendre
        sign = -1.0f;                 // sign from (NR3 6.7.10)
        ny_0_0 = 1.0f;
        ny_m_m = ny_0_0;
        fact2=1.0f;
        // init _pmm, pmm and pmmp1
        _pmm=1.0f;
        pmm = sqrt(1.0f/(4.0f*PI));
        pmmp1 = x*sqrt(3.0f)*pmm;    // l=l+1 (strange notation in NR3..)
        // various forward declarations
        float fac0,fac1,fac2,fac3,fac5,fac6,fac7,fac8, xi, accum_fac;
        float N_lm,K_nl,A_nl_t,C_n,C_n_prime,phi_nl_t,d_phi_nl_t_dr;
        float term1,term2,fac9,term;
        float C_lm,D_lm,E_lm,F_lm;

        int counter = 0;
        // sum up the coefficients
        for(int m = 0; m <= maxl; m++)
        {
            // init inverse renormalization (l)
            ny_l_m = ny_m_m;
            // init full legendre (NR3 6.7.8, untilded) (l)
            inv_renorm = (pi_4)/(2*m+1.0f);
            inv_renorm *= ny_l_m;
            inv_renorm = sqrt(inv_renorm);
            P_m_m = pmm * inv_renorm;
            P_l_m = P_m_m;
            // init deriv of legendre (l)
            DP_m_m = (x*m*P_m_m)/(x*x-1.0f);
            DP_l_m = DP_m_m;

            /*if(p == 323451 && counter == 0)
            {
                buggyval[0] = (x*x-1.0f);
                buggyval[1] = cos(th);
                buggyval[2] = th;
                counter++;
            }*/

            // init theta (l)
            theta_l_m = theta_m_m;
            // other stuff for the inner loop (l-loop)
            oldfact = sqrt(2.0f*m+3.0f);

            for (int l = m; l <= maxl; l++)
            {
                // compute the coefficients C_lm, D_lm, E_lm, F_lm
                C_lm=0.0f, D_lm=0.0f, E_lm=0.0f, F_lm=0.0f;

                // constant before float-sum
                N_lm = ( (2.0f*l + 1.0f)/(4.0f*PI) ) * theta_l_m;
                if(m != 0)				// TODO: faster check?
                {                       // TODO: omit this on the GPU
                    N_lm *= 2.0f;
                }
                // first part of float-sum (3.17) (sum over n)
                // precomputed constants for n-loop
                fac0 = (l + 1.0f)*(2.0f*l + 1.0f);
                fac1 = (4.0f*PI)/powf(2.0f,8.0f*l + 6.0f);
                //float fac2 = gamma2_cache[l]*gamma2_cache[l];
                fac2 = gamma2_constants[l]*gamma2_constants[l];
                fac3 = sqrt(4.0f*PI);
                fac5 = - ( powf(r, l)/powf(1 + r,2.0f*l+1.0f) ) * fac3;
                fac6 = (l/r) - ( (2.0f*l+1.0f)/(1.0f+r) );
                fac7 = ( (4.0f*(2.0f*l + 1.5f))/((1.0f+r)*(1.0f+r)));
                fac8 = fac3 * (-1.0f) * powf(r,l)/(powf(1+r,2.0f*l+1.0f));
                xi = (r - 1.0f) / (r + 1.0f);
                accum_fac = 1.0f;                             // accumulator for factorial

                for(int n = 0; n <= maxn; n++)
                {
                    K_nl = 0.5f*n*(n + 4.0f*l + 3.0f) + fac0;    // (2.23)
                    A_nl_t = accum_fac*(n + 2.0f*l + 1.5f ) * fac2;    // (2.31)
                    //A_nl_t /= -K_nl * fac1 * gamma1_cache[_map_gamma1_table(n,l)];
                    A_nl_t /= -K_nl * fac1 * gamma1_constants[_map_gamma1_table(n,l)];
                    C_n = _ultraspher(xi, n, 2.0f*l + 1.5f);
                    C_n_prime = _ultraspher(xi, n-1, 2.0f*l + 2.5f); // derivative of Gegenbauer polynomial
                    if(n == 0)
                    {
                        C_n_prime = 0;
                    }
                    phi_nl_t = fac5 * C_n;
                    d_phi_nl_t_dr = 0.0f;
                    term1 = fac6;                                     // (3.26)
                    term1 *= phi_nl_t;
                    term2 = fac7 * C_n_prime * fac8;                  // (3.26)
                    d_phi_nl_t_dr = term1 + term2;

                    fac9 = N_lm*A_nl_t;
                    C_lm += fac9 * phi_nl_t * x_constants[_map_x_table(n,0,l,m)];
                    D_lm += fac9 * phi_nl_t * x_constants[_map_x_table(n,1,l,m)];
                    E_lm += fac9 * d_phi_nl_t_dr * x_constants[_map_x_table(n,0,l,m)];
                    F_lm += fac9 * d_phi_nl_t_dr * x_constants[_map_x_table(n,1,l,m)];

                    // update the recurrence relations
                    accum_fac *= (n+1);                                     // update factorial accumulator
                }

                // now that we have the coefficients, use them to compute
                // another summand of the accelerations and sum up
                term = P_l_m * ( E_lm*cos(m*phi) + F_lm*sin(m*phi) );
                a_r += term;
                a_th += DP_l_m * (-sin(th)) * ( C_lm*cos(m*phi) + D_lm*sin(m*phi) );
                                                                            // the second factor here is the inner
                                                                            // derivative of the legendre poly
                a_phi += m*P_l_m * (D_lm*cos(m*phi) - C_lm*sin(m*phi));

                // update inverse renormalization factor
                ny_l_m *= ((l+m+1.0f)/(l-m+1.0f) );
                // update legendre
                ll = l + 2;
                fact=sqrt((4.0f*ll*ll-1.0f)/(ll*ll-m*m));
                pll=(x*pmmp1-pmm/oldfact)*fact;
                oldfact=fact;
                pmm=pmmp1;
                pmmp1=pll;
                // update P_l_m
                inv_renorm = (pi_4)/(2*(l+1)+1.0);
                inv_renorm *= ny_l_m;
                inv_renorm = sqrt(inv_renorm);
                P_lm1_m = P_l_m;                    // previous value needed for
                                                    // recursion of the derivative
                P_l_m = pmm * inv_renorm;
                // update derivative of legendre
                DP_l_m = 1.0/(x*x-1.0) * ( (l+1)*x*P_l_m - (l+1+m)*P_lm1_m );
                // update theta
                theta_l_m *= (l-m+1.0f)/(l+m+1.0f);
            }

            // update _pmm (without prefactor, see (6.7.10))
            _pmm *= omx2*fact2/(fact2+1.0f);
            fact2 += 2.0f;
            // update pmm (with prefactor, see (6.7.10))
            new_m = m+1;
            pmm=sqrt((2.0f*new_m+1.0f)*_pmm/(pi_4));
            pmm *= sign;
            sign *= -1.0f;
            // update pmmp1
            pmmp1=x*sqrt(2.0f*new_m+3.0f)*pmm;
            oldfact = sqrt(2.0f*new_m+3.0f);
            // update inverse renormalization factor
            ny_m_m *= (2*new_m-1)*(2*new_m);
            // update theta
            theta_m_m *= 1.0f/((2.0f*m+1.0f)*(2.0f*m+2.0f));
        }

        // multiply acceleration components with spherical gradient perfactors and negative sign
        // (see Hernquist 3.21 - 3.23)
        a_r *= -(1.0f);
        a_th *= -1.0f/r;
        a_phi *= -1.0f/(r*(sin(th)));

        // transform acceleration components to cartesian coords
        a_x = sin(th)*cos(phi)*a_r + cos(th)*cos(phi)*a_th - sin(phi)*a_phi;
        a_y = sin(th)*sin(phi)*a_r + cos(th)*sin(phi)*a_th + cos(phi)*a_phi;
        a_z = cos(th)*a_r - sin(th)*a_th;

        /*if(p == 323451)
        {
            buggyval[0] = a_r;
            buggyval[1] = a_th;
            buggyval[2] = a_phi;
            buggyval[3] = -1.0f/r;
            //buggyval[4] = a_r;
        }*/

        // store the accelerations
        accels[p*3] = a_x;
        accels[p*3+1] = a_y;
        accels[p*3+2] = a_z;

        // step 2: move to the next grid
        grid = grid + 1;
        p += ACCEL_GRID_DIM_X*ACCEL_BLOCK_DIM_X;

        __syncthreads();
    }


  /*  __syncthreads();
    if(tid == 0 && block == 0)
    {
        cuPrintf("buggy: %f\n", buggyval);
    }*/

}

__global__ void debug_kernel()
{
    for(int i = 0 ; i < 10; i++)
    {
//        cuPrintf("debug %d: %f\n", i, buggyval[i]);
    }
}

__device__ float _ultraspher(float xi, int n, float alpha)
{
    float C_nm1 = 1.0;
    if(n == 0)
        return C_nm1;
    float C_n = 2.0*alpha*xi;
    if(n == 1)
        return C_n;

    float C_m;
    for(int m = 2; m <= n; m++)
    {
        C_m = (1.0/m) * (2.0*((m-1.0)+alpha)*xi*C_n - ((m-1.0)+2.0*alpha-1.0)*C_nm1 );
        C_nm1 = C_n;
        C_n = C_m;
    }

    return C_m;
}

void reduce_list(float *data, int nrows, int size, float* gpu_result)
{
    // configure the kernel launch
    int maxThreads = 256;  // number of threads per block
    int whichKernel = 6;
    int maxBlocks = 64;
    int numBlocks = 0;
    int numThreads = 0;
    getNumBlocksAndThreads(whichKernel, size, maxBlocks, maxThreads, numBlocks, numThreads);

    //printf("num blocks = %d\n", numBlocks);

    // allocate memory for the results
    float* d_odata = NULL;
    float* h_odata = (float*) malloc(numBlocks*sizeof(float));
    cutilSafeCallNoSync( hipMalloc((void**) &d_odata, numBlocks*sizeof(float)) );

    // reduce each of the rows seperately
    for (int r = 0; r < nrows; ++r)
    {
        int row_sz = sizeof(float)*size;
        float* row = (float*)((char*)data + r * row_sz);
        hipDeviceSynchronize();
        reduce<float>(size, numThreads, numBlocks, whichKernel, row, d_odata);
        hipDeviceSynchronize();
        cutilSafeCallNoSync( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
        hipDeviceSynchronize();
        // reduce residual terms of the individual blocks on the CPU
        gpu_result[r] = 0.0;
        for(int i=0; i<numBlocks; i++)
        {
            //if(_debug0) printf("row %d, term %d: %f\n", r, i, h_odata[i]);
            gpu_result[r] += h_odata[i];
        }
    }

    // cleanup
    hipFree(d_odata);
}

extern "C"
bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

unsigned int nextPow2( unsigned int x )
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
    if (whichKernel < 3)
    {
        threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
        blocks = (n + threads - 1) / threads;
    }
    else
    {
        threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
        blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }


    if (whichKernel == 6)
        blocks = MIN(maxBlocks, blocks);
}

__global__ void _print_randomvals(float *rvals, int nparticles)
{
    //cuPrintf("randomvals:\n");

    for(int i = 0; i < nparticles; i++)
    {
        //cuPrintf("%f ", rvals[i]) ;
    }
    //cuPrintf("\n");
}

__global__ void _print_particles(float *particles, int nparticles, int dofs)
{
    //cuPrintf("particles:\n");

    for(int i = 0; i < nparticles; i++)
    {
        //cuPrintf("%d: %f %f %f %f\n", i, particles[i*dofs], particles[i*dofs+1], particles[i*dofs+2], particles[i*dofs+3]) ;
    }
    //cuPrintf("\n");
}

__device__ void _print_cache(float *cache)
{
    for(int n = 0; n < BLOCK_DIM_Y; n++)
    {
        //cuPrintf("n=%d: ", n);
        for(int p = 0; p < BLOCK_DIM_X; p++)
        {
            for(int x = 0; x < 2; x++)
            {
                for(int l = 0; l <= MAX_L; l++)
                {
                    for (int m = 0; m <= l; m++)
                    {
                        int cache_idx = _map_cache(n,p,x,l,m);
                        float val = cache[cache_idx];
                        ////cuPrintf("%.10f ", val);
                        //cuPrintf("%.4f ", val);
                    }
                }
                //cuPrintf(",");
            }
            //cuPrintf("\t;\t");
        }
        //cuPrintf("\n");
    }
}

__global__ void _print_list_all(float *list_all)
{
    //cuPrintf("all-list:\n");
    for(int grid = 0; grid < NUM_GRIDS; grid++)
    {
        //cuPrintf("grid %d:\n", grid);

        for(int n = 0; n < BLOCK_DIM_Y; n++)
        {
            //cuPrintf("n=%d: ", n);
            for(int block = 0; block < GRID_DIM_X; block++)
            {
                for(int x = 0; x < 2; x++)
                {
                    for(int l = 0; l <= MAX_L; l++)
                    {
                        for (int m = 0; m <= l; m++)
                        {
                            int list_all_idx = _map_list_all(grid, block, n, x, l, m);
                            float val = list_all[list_all_idx];
                            //cuPrintf("%.10f ", val);
                        }
                    }
                    ////cuPrintf(",");
                }
                //cuPrintf("\t;\t");
            }
            //cuPrintf("\n");
        }
        //cuPrintf("\n");
    }
}

__global__ void print_cuda_array(float *arr, int len, int whichrow)
{
    //cuPrintf("row %d: ", whichrow);
    for(int i = 0; i < len; i++)
    {
        //cuPrintf("%f ", arr[i]);
    }
    //cuPrintf("\n");
}

void shuffle_tables(float *table, float x1_table[MAX_N+1][MAX_L+1][MAX_L+1], float x2_table[MAX_N+1][MAX_L+1][MAX_L+1])
{
    for(int n = 0; n < BLOCK_DIM_Y; n++)
    {
        for(int l = 0; l <= MAX_L; l++)
        {
            for (int m = 0; m <= l; m++)
            {
                int x1_idx = map_x_table(n, 0, l, m);
                int x2_idx = map_x_table(n, 1, l, m);
                x1_table[n][l][m] = table[x1_idx];
                x2_table[n][l][m] = table[x2_idx];
            }
        }
    }
}

void compute_x_tables_GPU(float *particles, int nparticles,
                          float x1_table[MAX_N+1][MAX_L+1][MAX_L+1], float x2_table[MAX_N+1][MAX_L+1][MAX_L+1])
{
    // launch configuration
    int blocksPerGrid = GRID_DIM_X;						// for maximal performance
    dim3 threadsPerBlock(BLOCK_DIM_X, BLOCK_DIM_Y);		// (particlesPerThread, N_n)

    // allocate memory
    if(gmem_x_table_is_initialised == false)
    {
		cutilSafeCall( hipMalloc((void **)&d_list_all, LIST_ALL_SZ) );
		cutilSafeCall( hipMalloc((void **)&d_particles, PARTICLES_SZ) );
        gmem_x_table_is_initialised = true;
    }

    // init the timer
#ifdef GPU_TIMING
    hipEvent_t start, stop;
    float time_all_list;
    float time_reduce_list;
    float time_memcpy;
    float time_shuffle;
    float time_all;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

	// init the printer
#ifdef GPU_PRINT
	cudaPrintfInit();
#endif

    // copy data to device memory
#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    cutilSafeCall( hipMemcpy(d_particles, particles, PARTICLES_SZ, hipMemcpyHostToDevice) );
    cutilSafeCall( hipDeviceSynchronize() );
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_memcpy, start, stop);
#endif

    // perform the all-list kernel
#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    _compute_all_list<<< blocksPerGrid, threadsPerBlock>>>(d_particles, nparticles, d_list_all);
    cutilSafeCall( hipDeviceSynchronize() );
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_all_list, start, stop);
#endif

    // perform the reduce_list kernel
#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    float table[X_TABLE_ENTRIES];
    reduce_list(d_list_all, X_TABLE_ENTRIES, GRID_DIM_X*NUM_GRIDS, table);
    cutilSafeCall( hipDeviceSynchronize() );
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_reduce_list, start, stop);
#endif

    // shuffle the table array to the x1_table and x2_table parameters
#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    shuffle_tables(table, x1_table, x2_table);
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_shuffle, start, stop);
#endif

#ifdef GPU_TIMING
    time_all = time_all_list + time_reduce_list + time_memcpy + time_shuffle;
    printf("X-tables GPU timing results : ");
    printf("all-list: %f ms, reduce: %f ms, mem-copy: %f ms, shuffle: %f ms, all: %f ms\n", time_all_list, time_reduce_list, time_memcpy, time_shuffle, time_all);
#endif

    // cleanup the timer
#ifdef GPU_TIMING
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

    // cleanup the printer
#ifdef GPU_PRINT
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

}

void accel_gravity_scf_gpu(float *particles, int nparticles,
                          float x1_table[MAX_N+1][MAX_L+1][MAX_L+1], float x2_table[MAX_N+1][MAX_L+1][MAX_L+1],
                          float gamma1_table[MAX_N+1][MAX_L+1], float gamma2_table[MAX_L+1],
                          float *accels)
{
    // launch configuration
    int blocksPerGrid = ACCEL_GRID_DIM_X;				// for maximal performance
    int threadsPerBlock = ACCEL_BLOCK_DIM_X;              		// (particlesPerThread, N_n)

    // allocate memory
    if(gmem_accel_is_initialised == false)
    {
        cutilSafeCall( hipMalloc((void **)&d_particles, PARTICLES_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_accels, ACCELS_SZ) );
        gmem_accel_is_initialised = true;
    }

    // init the timer
#ifdef GPU_TIMING
    hipEvent_t start, stop;
    float time_accel;
    float time_memcpy;
    float time_all;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

	// init the printer
#ifdef GPU_PRINT
	cudaPrintfInit();
#endif

#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    // shuffle the non-linear host variables to linear temporary version which
    // can be directly copied with hipMemcpy
    float x_table_temp[X_TABLE_ENTRIES];
    float gamma1_table_temp[GAMMA1_TABLE_ENTRIES];
    for(int n = 0 ; n <= MAX_N; n++)    //X-tables
    {
        for(int l = 0; l <= MAX_L; l++)
        {
            for(int m=0; m <= l; m++)
            {
                int idx1 = map_x_table(n, 0, l, m);
                int idx2 = map_x_table(n, 1, l, m);
                x_table_temp[idx1] = x1_table[n][l][m];
                x_table_temp[idx2] = x2_table[n][l][m];
            }
        }
    }
    for(int n = 0; n <= MAX_N; n++)
    {
        for(int l = 0 ; l <= MAX_L; l++)
        {
            int idx = map_gamma1_table(n, l);
            gamma1_table_temp[idx] = gamma1_table[n][l];
        }
    }

    // copy data to device memory
    cutilSafeCall( hipMemcpy(d_particles, particles, PARTICLES_SZ, hipMemcpyHostToDevice) );
    cutilSafeCall( hipDeviceSynchronize() );

    // copy data to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(x_constants), x_table_temp, X_TABLE_SZ);
    hipMemcpyToSymbol(HIP_SYMBOL(gamma1_constants), gamma1_table_temp, GAMMA1_TABLE_SZ);
    hipMemcpyToSymbol(HIP_SYMBOL(gamma2_constants), gamma2_table, GAMMA2_TABLE_SZ);
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_memcpy, start, stop);
#endif

#ifdef GPU_TIMING
    hipEventRecord(start, 0);
#endif
    // perform the accel kernel
    printf("_accel() launch with blocksPerGrid=%d, threadsPerBlock=%d\n", blocksPerGrid,
            threadsPerBlock);
    _accel<<< blocksPerGrid, threadsPerBlock>>>(d_particles, nparticles, d_accels);
    cutilSafeCall( hipDeviceSynchronize() );
#ifdef GPU_TIMING
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_accel, start, stop);
#endif

    //debug_kernel<<<1,1>>>();

#ifdef GPU_TIMING
    time_all = time_accel + time_memcpy;
    printf("Acceleration GPU timing results : ");
    printf("accel: %f ms, mem-copy: %f ms, all: %f ms\n", time_accel, time_memcpy, time_all);
#endif

    // cleanup the timer
#ifdef GPU_TIMING
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

    // cleanup the printer
#ifdef GPU_PRINT
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

    // copy data to host
    cutilSafeCall( hipMemcpy(accels, d_accels, ACCELS_SZ, hipMemcpyDeviceToHost) );

}

void gpumem_cleanup()
{
    if(gmem_accel_is_initialised || gmem_x_table_is_initialised)
    {
        hipFree(d_particles);
    }
    if(gmem_accel_is_initialised)
    {
        hipFree(d_accels);
    }
    if(gmem_x_table_is_initialised)
    {
        hipFree(d_list_all);
    }
}

void gpu_test_kernel(float *particles, int nparticles)
{
    // launch configuration
    int blocksPerGrid = GRID_DIM_X;						// for maximal performance
    dim3 threadsPerBlock(BLOCK_DIM_X, BLOCK_DIM_Y);		// (particlesPerThread, N_n)

    // allocate memory
    if(gmem_x_table_is_initialised == false)
    {
		cutilSafeCall( hipMalloc((void **)&d_list_all, LIST_ALL_SZ) );
		cutilSafeCall( hipMalloc((void **)&d_particles, PARTICLES_SZ) );
        gmem_x_table_is_initialised = true;
    }

    // copy data to device memory
    cutilSafeCall( hipMemcpy(d_particles, particles, PARTICLES_SZ, hipMemcpyHostToDevice) );
    cutilSafeCall( hipDeviceSynchronize() );

}
